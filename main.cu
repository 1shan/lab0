#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"
#include "Timer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

//__device__ int a[3] = { 0, 1, 2 };

__global__ void SomeTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	 
	//if (idx < fsize && input_gpu[idx] != '\n') {
	//	input_gpu[idx] = '!';
	//}
	
	//**�N�r���������j�g**//
	if (idx < fsize && input_gpu[idx] != '\n'){
		if (input_gpu[idx] >= 'a' && input_gpu[idx] <= 'z'){
			input_gpu[idx] -= 32;
		}
	}

}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (!fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);//���NŪ�g��m�����ɧ�
	size_t fsize = ftell(fp); //�A�Ǧ^�ɮץثe��Ū�g��m(�ɧ�) ->�o���ɮת���
	fseek(fp, 0, SEEK_SET);//�A�q�Y�}�lŪ

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
	

	SomeTransform<<<102, 32>>>(input_gpu, fsize);  //2 gridDim.x, 32 blockDim.x

	puts(text_smem.get_cpu_ro());
	
	return 0;
}
